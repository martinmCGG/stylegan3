#include "hip/hip_runtime.h"
// Copyright (c) 2021, NVIDIA CORPORATION & AFFILIATES.  All rights reserved.
//
// NVIDIA CORPORATION and its licensors retain all intellectual property
// and proprietary rights in and to this software, related documentation
// and any modifications thereto.  Any use, reproduction, disclosure or
// distribution of this software and related documentation without an express
// license agreement from NVIDIA CORPORATION is strictly prohibited.

#include <c10/util/Half.h>
#include "upfirdn2d.h"

//------------------------------------------------------------------------
// Helpers.

template <class T> struct InternalType;
template <> struct InternalType<double>     { typedef double scalar_t; };
template <> struct InternalType<float>      { typedef float  scalar_t; };
template <> struct InternalType<c10::Half>  { typedef float  scalar_t; };

static __device__ __forceinline__ int floor_div(int a, int b)
{
    int t = 1 - a / b;
    return (a + t * b) / b - t;
}

//------------------------------------------------------------------------
// Generic CUDA implementation for large filters.

template <class T> __global__ void upfirdn2d_kernel_large(upfirdn2d_kernel_params p)
{
    typedef typename InternalType<T>::scalar_t scalar_t;

    // Calculate thread index.
    int minorBase = blockIdx.x * blockDim.x + threadIdx.x;
    int outY = minorBase / p.launchMinor;
    minorBase -= outY * p.launchMinor;
    int outXBase = blockIdx.y * p.loopX * blockDim.y + threadIdx.y;
    int majorBase = blockIdx.z * p.loopMajor;
    if (outXBase >= p.outSize.x | outY >= p.outSize.y | majorBase >= p.sizeMajor)
        return;

    // Setup Y receptive field.
    int midY = outY * p.down.y + p.up.y - 1 - p.pad0.y;
    int inY = min(max(floor_div(midY, p.up.y), 0), p.inSize.y);
    int h = min(max(floor_div(midY + p.filterSize.y, p.up.y), 0), p.inSize.y) - inY;
    int filterY = midY + p.filterSize.y - (inY + 1) * p.up.y;
    if (p.flip)
        filterY = p.filterSize.y - 1 - filterY;

    // Loop over major, minor, and X.
    for (int majorIdx = 0, major = majorBase; majorIdx < p.loopMajor & major < p.sizeMajor; majorIdx++, major++)
    for (int minorIdx = 0, minor = minorBase; minorIdx < p.loopMinor & minor < p.sizeMinor; minorIdx++, minor += p.launchMinor)
    {
        int nc = major * p.sizeMinor + minor;
        int n = nc / p.inSize.z;
        int c = nc - n * p.inSize.z;
        for (int loopX = 0, outX = outXBase; loopX < p.loopX & outX < p.outSize.x; loopX++, outX += blockDim.y)
        {
            // Setup X receptive field.
            int midX = outX * p.down.x + p.up.x - 1 - p.pad0.x;
            int inX = min(max(floor_div(midX, p.up.x), 0), p.inSize.x);
            int w = min(max(floor_div(midX + p.filterSize.x, p.up.x), 0), p.inSize.x) - inX;
            int filterX = midX + p.filterSize.x - (inX + 1) * p.up.x;
            if (p.flip)
                filterX = p.filterSize.x - 1 - filterX;

            // Initialize pointers.
            const T* xp = &((const T*)p.x)[inX * p.inStride.x + inY * p.inStride.y + c * p.inStride.z + n * p.inStride.w];
            const float* fp = &p.f[filterX * p.filterStride.x + filterY * p.filterStride.y];
            int filterStepX = ((p.flip) ? p.up.x : -p.up.x) * p.filterStride.x;
            int filterStepY = ((p.flip) ? p.up.y : -p.up.y) * p.filterStride.y;

            // Inner loop.
            scalar_t v = 0;
            for (int y = 0; y < h; y++)
            {
                for (int x = 0; x < w; x++)
                {
                    v += (scalar_t)(*xp) * (scalar_t)(*fp);
                    xp += p.inStride.x;
                    fp += filterStepX;
                }
                xp += p.inStride.y - w * p.inStride.x;
                fp += filterStepY - w * filterStepX;
            }

            // Store result.
            v *= p.gain;
            ((T*)p.y)[outX * p.outStride.x + outY * p.outStride.y + c * p.outStride.z + n * p.outStride.w] = (T)v;
        }
    }
}

//------------------------------------------------------------------------
// Specialized CUDA implementation for small filters.

template <class T, int upx, int upy, int downx, int downy, int filterW, int filterH, int tileOutW, int tileOutH, int loopMinor>
__global__ void upfirdn2d_kernel_small(upfirdn2d_kernel_params p)
{
    typedef typename InternalType<T>::scalar_t scalar_t;
    const int tileInW = ((tileOutW - 1) * downx + filterW - 1) / upx + 1;
    const int tileInH = ((tileOutH - 1) * downy + filterH - 1) / upy + 1;
    __shared__ volatile scalar_t sf[filterH][filterW];
    __shared__ volatile scalar_t sx[tileInH][tileInW][loopMinor];

    // Calculate tile index.
    int minorBase = blockIdx.x;
    int tileOutY = minorBase / p.launchMinor;
    minorBase -= tileOutY * p.launchMinor;
    minorBase *= loopMinor;
    tileOutY *= tileOutH;
    int tileOutXBase = blockIdx.y * p.loopX * tileOutW;
    int majorBase = blockIdx.z * p.loopMajor;
    if (tileOutXBase >= p.outSize.x | tileOutY >= p.outSize.y | majorBase >= p.sizeMajor)
        return;

    // Load filter (flipped).
    for (int tapIdx = threadIdx.x; tapIdx < filterH * filterW; tapIdx += blockDim.x)
    {
        int fy = tapIdx / filterW;
        int fx = tapIdx - fy * filterW;
        scalar_t v = 0;
        if (fx < p.filterSize.x & fy < p.filterSize.y)
        {
            int ffx = (p.flip) ? fx : p.filterSize.x - 1 - fx;
            int ffy = (p.flip) ? fy : p.filterSize.y - 1 - fy;
            v = (scalar_t)p.f[ffx * p.filterStride.x + ffy * p.filterStride.y];
        }
        sf[fy][fx] = v;
    }

    // Loop over major and X.
    for (int majorIdx = 0, major = majorBase; majorIdx < p.loopMajor & major < p.sizeMajor; majorIdx++, major++)
    {
        int baseNC = major * p.sizeMinor + minorBase;
        int n = baseNC / p.inSize.z;
        int baseC = baseNC - n * p.inSize.z;
        for (int loopX = 0, tileOutX = tileOutXBase; loopX < p.loopX & tileOutX < p.outSize.x; loopX++, tileOutX += tileOutW)
        {
            // Load input pixels.
            int tileMidX = tileOutX * downx + upx - 1 - p.pad0.x;
            int tileMidY = tileOutY * downy + upy - 1 - p.pad0.y;
            int tileInX = floor_div(tileMidX, upx);
            int tileInY = floor_div(tileMidY, upy);
            __syncthreads();
            for (int inIdx = threadIdx.x; inIdx < tileInH * tileInW * loopMinor; inIdx += blockDim.x)
            {
                int relC = inIdx;
                int relInX = relC / loopMinor;
                int relInY = relInX / tileInW;
                relC -= relInX * loopMinor;
                relInX -= relInY * tileInW;
                int c = baseC + relC;
                int inX = tileInX + relInX;
                int inY = tileInY + relInY;
                scalar_t v = 0;
                if (inX >= 0 & inY >= 0 & inX < p.inSize.x & inY < p.inSize.y & c < p.inSize.z)
                    v = (scalar_t)((const T*)p.x)[inX * p.inStride.x + inY * p.inStride.y + c * p.inStride.z + n * p.inStride.w];
                sx[relInY][relInX][relC] = v;
            }

            // Loop over output pixels.
            __syncthreads();
            for (int outIdx = threadIdx.x; outIdx < tileOutH * tileOutW * loopMinor; outIdx += blockDim.x)
            {
                int relC = outIdx;
                int relOutX = relC / loopMinor;
                int relOutY = relOutX / tileOutW;
                relC -= relOutX * loopMinor;
                relOutX -= relOutY * tileOutW;
                int c = baseC + relC;
                int outX = tileOutX + relOutX;
                int outY = tileOutY + relOutY;

                // Setup receptive field.
                int midX = tileMidX + relOutX * downx;
                int midY = tileMidY + relOutY * downy;
                int inX = floor_div(midX, upx);
                int inY = floor_div(midY, upy);
                int relInX = inX - tileInX;
                int relInY = inY - tileInY;
                int filterX = (inX + 1) * upx - midX - 1; // flipped
                int filterY = (inY + 1) * upy - midY - 1; // flipped

                // Inner loop.
                if (outX < p.outSize.x & outY < p.outSize.y & c < p.outSize.z)
                {
                    scalar_t v = 0;
                    #pragma unroll
                    for (int y = 0; y < filterH / upy; y++)
                        #pragma unroll
                        for (int x = 0; x < filterW / upx; x++)
                            v += sx[relInY + y][relInX + x][relC] * sf[filterY + y * upy][filterX + x * upx];
                    v *= p.gain;
                    ((T*)p.y)[outX * p.outStride.x + outY * p.outStride.y + c * p.outStride.z + n * p.outStride.w] = (T)v;
                }
            }
        }
    }
}


//------------------------------------------------------------------------
// Template specializations.

#define SPEC_with_type(f, ...) \
    template __global__ void f<__VA_ARGS__>(upfirdn2d_kernel_params p);

#define SPEC(f, ...) \
    SPEC_with_type(f, double, __VA_ARGS__) \
    SPEC_with_type(f, float, __VA_ARGS__) \
    SPEC_with_type(f, c10::Half, __VA_ARGS__)

// "large" kernel specializations
template __global__ void upfirdn2d_kernel_large<double>(upfirdn2d_kernel_params p);
template __global__ void upfirdn2d_kernel_large<float>(upfirdn2d_kernel_params p);
template __global__ void upfirdn2d_kernel_large<c10::Half>(upfirdn2d_kernel_params p);

// Instead of writing full specializations for all the variants of the "small" kernel and it data type (~300 difficult-to-read lines like this):
//   template __global__ void upfirdn2d_kernel_small<double, 1, 1, 1, 4, 1, 48, 32, 8, 1>(upfirdn2d_kernel_params p);
//   template __global__ void upfirdn2d_kernel_small<float, 1, 1, 1, 4, 1, 48, 32, 8, 1>(upfirdn2d_kernel_params p);
//   template __global__ void upfirdn2d_kernel_small<c10::Half, 1, 1, 1, 4, 1, 48, 32, 8, 1>(upfirdn2d_kernel_params p);
// we can just write "SPEC(kernel_name, params)", e.g. "SPEC(upfirdn2d_kernel_small, 1, 1, 1, 4, 1, 48, 32, 8, 1)" to make specializations for all types of one kernel variation with a single line.
// These lines can be generated automatically from the file they are called from, using the following command:
//   grep 'upfirdn2d_kernel_\(small\|large\)<T, .*>(p)' torch_utils/ops/upfirdn2d.cpp | sed 's/.*\(upfirdn2d_kernel_\(small\|large\)\)<T,\(.*\)>.*/SPEC(\1,\3)/'

SPEC(upfirdn2d_kernel_small, 1,1, 1,4, 1,32, 1,32,8)
SPEC(upfirdn2d_kernel_small, 1,1, 1,4, 1,48, 1,32,8)
SPEC(upfirdn2d_kernel_small, 1,1, 1,4, 1,32, 32,8,1)
SPEC(upfirdn2d_kernel_small, 1,1, 1,4, 1,48, 32,8,1)
SPEC(upfirdn2d_kernel_small, 1,1, 4,1, 32,1, 32,1,8)
SPEC(upfirdn2d_kernel_small, 1,1, 4,1, 48,1, 32,1,8)
SPEC(upfirdn2d_kernel_small, 1,1, 4,1, 32,1, 32,8,1)
SPEC(upfirdn2d_kernel_small, 1,1, 4,1, 48,1, 32,8,1)
SPEC(upfirdn2d_kernel_small, 1,4, 1,1, 1,32, 1,128,16)
SPEC(upfirdn2d_kernel_small, 1,4, 1,1, 1,48, 1,128,16)
SPEC(upfirdn2d_kernel_small, 1,4, 1,1, 1,32, 32,32,1)
SPEC(upfirdn2d_kernel_small, 1,4, 1,1, 1,48, 32,32,1)
SPEC(upfirdn2d_kernel_small, 4,1, 1,1, 32,1, 128,1,16)
SPEC(upfirdn2d_kernel_small, 4,1, 1,1, 48,1, 128,1,16)
SPEC(upfirdn2d_kernel_small, 4,1, 1,1, 32,1, 128,8,1)
SPEC(upfirdn2d_kernel_small, 4,1, 1,1, 48,1, 128,8,1)
SPEC(upfirdn2d_kernel_small, 4,4, 1,1, 32,32, 32,32,1)
SPEC(upfirdn2d_kernel_small, 4,4, 1,1, 48,48, 32,32,1)
SPEC(upfirdn2d_kernel_small, 4,4, 1,1, 32,32, 64,32,1)
SPEC(upfirdn2d_kernel_small, 4,4, 1,1, 48,48, 64,32,1)
SPEC(upfirdn2d_kernel_small, 1,1, 1,2, 1,8,  1,64,8)
SPEC(upfirdn2d_kernel_small, 1,1, 1,2, 1,16, 1,64,8)
SPEC(upfirdn2d_kernel_small, 1,1, 1,2, 1,24, 1,64,8)
SPEC(upfirdn2d_kernel_small, 1,1, 1,2, 1,8,  32,16,1)
SPEC(upfirdn2d_kernel_small, 1,1, 1,2, 1,16, 32,16,1)
SPEC(upfirdn2d_kernel_small, 1,1, 1,2, 1,24, 32,16,1)
SPEC(upfirdn2d_kernel_small, 1,1, 2,1, 8,1,  64,1,8)
SPEC(upfirdn2d_kernel_small, 1,1, 2,1, 16,1, 64,1,8)
SPEC(upfirdn2d_kernel_small, 1,1, 2,1, 24,1, 64,1,8)
SPEC(upfirdn2d_kernel_small, 1,1, 2,1, 8,1,  64,8,1)
SPEC(upfirdn2d_kernel_small, 1,1, 2,1, 16,1, 64,8,1)
SPEC(upfirdn2d_kernel_small, 1,1, 2,1, 24,1, 64,8,1)
SPEC(upfirdn2d_kernel_small, 1,1, 2,2, 2,2,   8,8,8)
SPEC(upfirdn2d_kernel_small, 1,1, 2,2, 4,4,   8,8,8)
SPEC(upfirdn2d_kernel_small, 1,1, 2,2, 6,6,   8,8,8)
SPEC(upfirdn2d_kernel_small, 1,1, 2,2, 8,8,   8,8,8)
SPEC(upfirdn2d_kernel_small, 1,1, 2,2, 16,16, 16,16,1)
SPEC(upfirdn2d_kernel_small, 1,1, 2,2, 24,24, 16,16,1)
SPEC(upfirdn2d_kernel_small, 1,1, 2,2, 2,2,   32,8,1)
SPEC(upfirdn2d_kernel_small, 1,1, 2,2, 4,4,   32,8,1)
SPEC(upfirdn2d_kernel_small, 1,1, 2,2, 6,6,   32,8,1)
SPEC(upfirdn2d_kernel_small, 1,1, 2,2, 8,8,   32,8,1)
SPEC(upfirdn2d_kernel_small, 1,1, 2,2, 16,16, 32,16,1)
SPEC(upfirdn2d_kernel_small, 1,1, 2,2, 24,24, 32,16,1)
SPEC(upfirdn2d_kernel_small, 1,2, 1,1, 1,8,  1,128,16)
SPEC(upfirdn2d_kernel_small, 1,2, 1,1, 1,16, 1,128,16)
SPEC(upfirdn2d_kernel_small, 1,2, 1,1, 1,24, 1,128,16)
SPEC(upfirdn2d_kernel_small, 1,2, 1,1, 1,8,  32,32,1)
SPEC(upfirdn2d_kernel_small, 1,2, 1,1, 1,16, 32,32,1)
SPEC(upfirdn2d_kernel_small, 1,2, 1,1, 1,24, 32,32,1)
SPEC(upfirdn2d_kernel_small, 2,1, 1,1, 8,1,  128,1,16)
SPEC(upfirdn2d_kernel_small, 2,1, 1,1, 16,1, 128,1,16)
SPEC(upfirdn2d_kernel_small, 2,1, 1,1, 24,1, 128,1,16)
SPEC(upfirdn2d_kernel_small, 2,1, 1,1, 8,1,  128,8,1)
SPEC(upfirdn2d_kernel_small, 2,1, 1,1, 16,1, 128,8,1)
SPEC(upfirdn2d_kernel_small, 2,1, 1,1, 24,1, 128,8,1)
SPEC(upfirdn2d_kernel_small, 2,2, 1,1, 2,2,   16,16,8)
SPEC(upfirdn2d_kernel_small, 2,2, 1,1, 4,4,   16,16,8)
SPEC(upfirdn2d_kernel_small, 2,2, 1,1, 6,6,   16,16,8)
SPEC(upfirdn2d_kernel_small, 2,2, 1,1, 8,8,   16,16,8)
SPEC(upfirdn2d_kernel_small, 2,2, 1,1, 16,16, 32,32,1)
SPEC(upfirdn2d_kernel_small, 2,2, 1,1, 24,24, 32,32,1)
SPEC(upfirdn2d_kernel_small, 2,2, 1,1, 2,2,   64,16,1)
SPEC(upfirdn2d_kernel_small, 2,2, 1,1, 4,4,   64,16,1)
SPEC(upfirdn2d_kernel_small, 2,2, 1,1, 6,6,   64,16,1)
SPEC(upfirdn2d_kernel_small, 2,2, 1,1, 8,8,   64,16,1)
SPEC(upfirdn2d_kernel_small, 2,2, 1,1, 16,16, 64,32,1)
SPEC(upfirdn2d_kernel_small, 2,2, 1,1, 24,24, 64,32,1)
SPEC(upfirdn2d_kernel_small, 1,1, 1,1, 1,8,   1,128,16)
SPEC(upfirdn2d_kernel_small, 1,1, 1,1, 1,16,  1,128,16)
SPEC(upfirdn2d_kernel_small, 1,1, 1,1, 1,24,  1,128,16)
SPEC(upfirdn2d_kernel_small, 1,1, 1,1, 8,1,   128,1,16)
SPEC(upfirdn2d_kernel_small, 1,1, 1,1, 16,1,  128,1,16)
SPEC(upfirdn2d_kernel_small, 1,1, 1,1, 24,1,  128,1,16)
SPEC(upfirdn2d_kernel_small, 1,1, 1,1, 3,3,   16,16,8)
SPEC(upfirdn2d_kernel_small, 1,1, 1,1, 4,4,   16,16,8)
SPEC(upfirdn2d_kernel_small, 1,1, 1,1, 5,5,   16,16,8)
SPEC(upfirdn2d_kernel_small, 1,1, 1,1, 6,6,   16,16,8)
SPEC(upfirdn2d_kernel_small, 1,1, 1,1, 7,7,   16,16,8)
SPEC(upfirdn2d_kernel_small, 1,1, 1,1, 16,16, 32,32,1)
SPEC(upfirdn2d_kernel_small, 1,1, 1,1, 24,24, 32,32,1)
SPEC(upfirdn2d_kernel_small, 1,1, 1,1, 1,8,   32,32,1)
SPEC(upfirdn2d_kernel_small, 1,1, 1,1, 1,16,  32,32,1)
SPEC(upfirdn2d_kernel_small, 1,1, 1,1, 1,24,  32,32,1)
SPEC(upfirdn2d_kernel_small, 1,1, 1,1, 8,1,   128,8,1)
SPEC(upfirdn2d_kernel_small, 1,1, 1,1, 16,1,  128,8,1)
SPEC(upfirdn2d_kernel_small, 1,1, 1,1, 24,1,  128,8,1)
SPEC(upfirdn2d_kernel_small, 1,1, 1,1, 3,3,   64,16,1)
SPEC(upfirdn2d_kernel_small, 1,1, 1,1, 4,4,   64,16,1)
SPEC(upfirdn2d_kernel_small, 1,1, 1,1, 5,5,   64,16,1)
SPEC(upfirdn2d_kernel_small, 1,1, 1,1, 6,6,   64,16,1)
SPEC(upfirdn2d_kernel_small, 1,1, 1,1, 7,7,   64,16,1)
SPEC(upfirdn2d_kernel_small, 1,1, 1,1, 16,16, 64,32,1)
SPEC(upfirdn2d_kernel_small, 1,1, 1,1, 24,24, 64,32,1)

//------------------------------------------------------------------------
